#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h" 
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>

#define MAX_THREADS_PER_BLOCK 1024

// Method 1 串行
// CPU Merge Recursive Call function
void merge(int* arr, int* temp, int left, int mid, int right) 
{
    int i = left;
    int j = mid + 1;
    int k = left;

    while (i <= mid && j <= right) 
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i <= mid)
        temp[k++] = arr[i++];

    while (j <= right)
        temp[k++] = arr[j++];

    for (int idx = left; idx <= right; ++idx)
        arr[idx] = temp[idx];
}

// CPU Implementation of Merge Sort
void mergeSortCPU(int* arr, int* temp, int left, int right) 
{
    if (left >= right)
        return;

    int mid = left + (right - left) / 2;

    mergeSortCPU(arr, temp, left, mid);
    mergeSortCPU(arr, temp, mid + 1, right);

    merge(arr, temp, left, mid, right);
}

// Method 2 并行
// Device function for recursive Merge
__device__ void Merge(int* arr, int* temp, int left, int middle, int right) 
{
    int i = left;
    int j = middle; 
    int k = left;

    while (i < middle && j < right) 
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i < middle)
        temp[k++] = arr[i++];
    while (j < right)
        temp[k++] = arr[j++];

    for (int x = left; x < right; x++)
        arr[x] = temp[x];
}

// GPU Kernel for Merge Sort
__global__ void MergeSortGPU(int* arr, int* temp, int n, int width) 
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int left = tid * width;
    int middle = left + width / 2;
    int right = left + width;

    if (left < n && middle < n) 
    {
        Merge(arr, temp, left, middle, right);
    }
}

// Method 3 共享内存并行
// 共享内存归并排序 - 每个块内进行归并排序
__global__ void sharedMemoryMergeSort(int* arr, int* temp, int n) 
{
    extern __shared__ int sharedArr[]; // 共享内存声明
    int local_tid = threadIdx.x; // 计算块内索引
    int block_start = blockIdx.x * blockDim.x; // 当前块的起始索引

    // 1. 加载数据到共享内存
    if (block_start + local_tid < n) 
    {
        sharedArr[local_tid] = arr[block_start + local_tid];
    }
    __syncthreads();
    // 2. 共享内存内部执行归并排序
    for (int width = 1; width < blockDim.x; width *= 2) {
        int left = local_tid * width * 2;
        int middle = left + width;
        int right = min(left + width * 2, blockDim.x);

        if (middle < blockDim.x) {
            Merge(sharedArr, temp, left, middle, right);
        }
        __syncthreads();
    }

    // 3. 写回全局内存
    // 计算 `validSize`（非零部分）
    int validSize = 0;
    for (int i = 0; i < blockDim.x; i++) {
        if (sharedArr[i] > 0) {
            validSize++;
        }
    }
    // 写回全局内存
    if (block_start + local_tid < n) {
        arr[block_start + local_tid] = sharedArr[local_tid + (blockDim.x - validSize)];
    }
}



// 功能函数
// Function to print array
void printArray(int* arr, int size) 
{
    for (int i = 0; i < size; ++i)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

// Automated function to check if array is sorted
bool isSorted(int* arr, int size) 
{
    for (int i = 1; i < size; ++i) 
    {
        if (arr[i] < arr[i - 1])
            return false;
    }
    return true;
}

// Function to check if given number is a power of 2
bool isPowerOfTwo(int num) 
{
    return num > 0 && (num & (num - 1)) == 0;
}


// MAIN PROGRAM
int main()
{   
    std::cout << "-----------------------------------------------" << std::endl;
    std::cout << "MERGE SORT IMPLEMENTATION" << std::endl;
    std::cout << "A Performance Comparison of These 4 Sorts in CPU vs GPU vs sharedMemory vs ？？？ " << std::endl;
    std::cout << "-----------------------------------------------" << std::endl;
    

    // 输入大小
    int size;
    std::cout << "\n\nEnter the size of the array. Must be a power of 2:\n ";
    std::cin>>size;
    // 判断是否为2的幂
    while (!isPowerOfTwo(size))
    {
        if (!isPowerOfTwo(size))
        {
            std::cout << "\nWrong Size, must be power of 2. Try again:\n ";
            std::cin>>size;
        }
        else
            break;
    }
    

    // 初始化数组
    //Create CPU based Arrays
    int* arr = new int[size];
    int* arr2 = new int[size];
    //int* arr3 = new int[size];？？？
    int* carr = new int[size];
    int* temp = new int[size];


    //Create GPU based arrays
    int* gpuArrmerge;
    int* gpuArrmerge2;
    //int* gpuArrmerge3; ？？？
    int* gpuTemp;
    int* gpuTemp2;

    // Initialize the array with random values
    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < size; ++i) 
    {
        arr[i] = rand() % 256 + 1;
        arr2[i] = arr[i];
        //arr3[i]
        carr[i] = arr[i];
    }

    //Print unsorted array 
    std::cout << "\n\nUnsorted array: ";
    if (size <= 256) 
    {
        printArray(arr, size);
    }
    else 
    {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }


    // 排序


    // CPU
    // 初始化时间变量
    // Initialize CPU clock counters
    clock_t startCPU, endCPU;
    // Time the CPU and call CPU Merge Sort
    startCPU = clock();
    mergeSortCPU(carr, temp, 0, size - 1);
    endCPU = clock();
    // Calculate Elapsed CPU time
    double millisecondsCPU = static_cast<double>(endCPU - startCPU) / (CLOCKS_PER_SEC / 1000.0);
    
    
    // GPU
    // 分配GPU内存
    // Allocate memory on GPU
    hipMalloc((void**)&gpuArrmerge, size * sizeof(int));
    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    // Copy the input array to GPU memory
    hipMemcpy(gpuArrmerge, arr, size * sizeof(int), hipMemcpyHostToDevice); 
    
    // 初始化时间变量
    // Perform GPU merge sort and measure time
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float millisecondsGPU = 0;
    
    // Set number of threads and blocks for kernel calls
    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
     //Call GPU Merge Kernel and time the run
    hipEventRecord(startGPU);
    for (int wid = 1; wid < size; wid *= 2)
    {
        MergeSortGPU << <threadsPerBlock, blocksPerGrid >> > (gpuArrmerge, gpuTemp, size, wid * 2);
    }
    hipEventRecord(stopGPU);
    
    // 传送结果
    //Transfer sorted array back to CPU
    hipMemcpy(arr, gpuArrmerge, size * sizeof(int), hipMemcpyDeviceToHost);
    //Calculate Elapsed GPU time
    hipEventSynchronize(stopGPU);
    hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);
    //End
    hipFree(gpuArrmerge);
    hipFree(gpuTemp);


    // 共享内存
    // 分配GPU内存
    // Allocate memory on GPU
    hipMalloc((void**)&gpuArrmerge2, size * sizeof(int));
    hipMalloc((void**)&gpuTemp2, size * sizeof(int));
    // Copy the input array to GPU memory
    hipMemcpy(gpuArrmerge2, arr2, size * sizeof(int), hipMemcpyHostToDevice);   
    
    // 初始化时间变量
    // Perform sharedMemory merge sort and measure time
    hipEvent_t startGPU2, stopGPU2; 
    hipEventCreate(&startGPU2);
    hipEventCreate(&stopGPU2);
    float millisecondsGPU2 = 0;  

    // 确定线程块大小
    int threadsPerBlock2 = 256; // 每个线程块的线程数
    int blocksPerGrid2 = (size + threadsPerBlock2 - 1) / threadsPerBlock2; // 计算线程块数量        
    // 计算共享内存的大小：每个线程块需要至少存储一个元素
    int sharedMemSize = threadsPerBlock2 * sizeof(int);
    // 记录开始时间
    hipEventRecord(startGPU2);
    // 启动归并排序内核
    sharedMemoryMergeSort << < blocksPerGrid2, threadsPerBlock2, sharedMemSize >> > (gpuArrmerge2, gpuTemp2, size);
    // 记录结束时间
    hipEventRecord(stopGPU2);

    // 传送结果
    //Transfer sorted array back to CPU
    hipMemcpy(arr2, gpuArrmerge2, size * sizeof(int), hipMemcpyDeviceToHost);
    //Calculate Elapsed GPU time
    hipEventSynchronize(stopGPU2);
    hipEventElapsedTime(&millisecondsGPU2, startGPU2, stopGPU2);
    //End
    hipFree(gpuArrmerge2);
    hipFree(gpuTemp2);
        


    // 输出结果
    // Display sorted CPU array
    std::cout << "\n\nSorted CPU array: ";
    if (size <= 256) 
    {
        printArray(carr, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    // Display sorted GPU array
    std::cout << "\n\nSorted GPU array: ";
    if (size <= 256) 
    {
        printArray(arr, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    // Display sorted sharedMemory array
    std::cout << "\n\nSorted sharedMemory array: ";
    if (size <= 256) 
    {
        printArray(arr2, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }
    
    //Run the array with the automated isSorted checker
    if (isSorted(carr, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED CPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;

    if (isSorted(arr, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED GPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;
    
    if (isSorted(arr2, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED GPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;

    //Print the time of the runs
    std::cout << "\n\nCPU Time: " << millisecondsCPU << " ms" << std::endl;
    std::cout << "GPU Time: " << millisecondsGPU << " ms" << std::endl;
    std::cout << "sharedMemory Time: " << millisecondsGPU2 << " ms" << std::endl;

    //Destroy all variables
    delete[] carr;
    delete[] arr;
    delete[] arr2;
    delete[] temp;

    std::cout << "\n------------------------------------------------------------------------------------\n||||| END. YOU MAY RUN THIS AGAIN |||||\n------------------------------------------------------------------------------------\n";
    return 0;
}